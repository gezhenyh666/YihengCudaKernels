
#include <hip/hip_runtime.h>
#include "../include/cuda.h"
#include "../include/cuda_runtime.h"
#include "../include/device_launch_parameters.h"
#include "../include/crt/device_functions.h"

const int KM = 2048;
const int KK = 2048;
const int KN = 2048;

const int KBm = 32;
const int KBk = 32;
const int KBn = 32;

/*
使用shared mem
*/
__global__ void MatrixMultiplyUseSharedMemV0(float* input_m, float* input_n, float* output, int m, int k, int n)
{
    __shared__ float ms[KBm][KBk];
    __shared__ float ns[KBk][KBn];
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    for (int k = 0; k < (KBk + KK - 1) / KBk; k++) {
        ms[threadIdx.y][threadIdx.x] = input_m[row_idx*KK+k*KBk+threadIdx.x];
        ns[threadIdx.y][threadIdx.x] = input_n[(k*KBk+threadIdx.y)*KN+col_idx];
        __syncthreads();

        for (int i = 0; i < KBk; i++) {
           sum += ms[threadIdx.y][i] * ns[i][threadIdx.x]; 
        }
        __syncthreads();
    }

    output[row_idx*KN+col_idx] = sum;
}