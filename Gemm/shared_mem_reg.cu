
#include <hip/hip_runtime.h>
#include "../include/cuda.h"
#include "../include/cuda_runtime.h"
#include "../include/device_launch_parameters.h"
#include "../include/crt/device_functions.h"

const int KM = 2048;
const int KK = 2048;
const int KN = 2048;

const int KBm = 128;
const int KBk = 8;
const int KBn = 128;

const int KTm = 8;
const int KTn = 8;

/*
使用shared mem + reg
*/
__global__ void MatrixMultiplyUseSharedMemReg(float* input_m, float* input_n, float* output, int m, int k, int n)
{
    __shared__ float ms[KBm][KBk];
    __shared__ float ns[KBk][KBn];

    float r_c[KTm][KTn] = {0};

    // tid表示在对应thread在block中的全局id
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    

}