
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/cuda.h"
#include "../include/cuda_runtime.h"
#include "../include/device_launch_parameters.h"
#include "../include/cuComplex.h"

__device__ float GetMaxValue(float a, float b)
{
    if (a < b) {
        return a;
    } else {
        return b;
    }
}
__global__ void SoftMax(float *input, float *output, int len)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        float max_value = input[idx];
        for (int i = 0; i < len; i++) {
            max_value = GetMaxValue(max_value, input[i]);
        }

        float sum = 0.0;
        for (int i = 0; i < len; i++) {
            sum += __expf(input[i]-max_value);
        }
        output[idx] = __expf(input[idx] - max_value) / sum;
    }
}
